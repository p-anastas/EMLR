#include <float.h>
#include <cstdio>
#include "cpu_utils.hpp"
#include "gpu_utils.hpp"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

gpu_timer_p gpu_timer_init() {
	gpu_timer_p timer = (gpu_timer_p)malloc(sizeof(struct gpu_timer));
	hipEventCreate(&timer->start);
	hipEventCreate(&timer->stop);
	return timer;
}

void gpu_timer_start(gpu_timer_p timer, hipStream_t stream) { hipEventRecord(timer->start, stream); }

void gpu_timer_stop(gpu_timer_p timer, hipStream_t stream) { hipEventRecord(timer->stop, stream); }

float gpu_timer_get(gpu_timer_p timer) {
	hipEventSynchronize(timer->stop);
	hipEventElapsedTime(&timer->ms, timer->start, timer->stop);
	return timer->ms;
}

const char *print_mem(mem_layout mem) {
	if (mem == ROW_MAJOR) return "Row major";
	else if (mem == COL_MAJOR) return "Col major";
	else return "ERROR";
}

const char *print_loc(short loc) {
	int dev_count;
	hipGetDeviceCount(&dev_count);
	if (loc == -2)  return "Host"; 
	else if (loc == -1 || loc == -3)  return "Pinned Host";
	else if (loc < dev_count) return "Device";
	else return "ERROR";
}

void cudaCheckErrors() {
	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess) printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

/// Allocate 'count' bytes of CUDA device memory (+errorcheck)
void *gpu_malloc(long long count) {
	void *ret;
	massert(hipMalloc(&ret, count) == hipSuccess,
          hipGetErrorString(hipGetLastError()));
	return ret;
}

/// Allocate 'count' bytes of CUDA host pinned memory (+errorcheck)
void *pin_malloc(long long count) {
	void *ret;
	massert(hipHostMalloc(&ret, count) == hipSuccess,
          hipGetErrorString(hipGetLastError()));
	return ret;
}

void vec_alloc(void ** ptr, long long N_bytes, int loc){
	int count = 666;
	hipGetDeviceCount(&count);

	if (-2 == loc) {
    		//fprintf(stderr, "Allocating %lld bytes to host...\n", N_bytes);
		*ptr = (void*) malloc(N_bytes);
	}
	else if (-1 == loc || -3 == loc) {
		//fprintf(stderr, "Allocating %lld bytes to pinned host...\n", N_bytes);
		*ptr = pin_malloc(N_bytes);

	} else if (loc >= count || loc < 0) error("vec_init: Invalid device");
	else {
		//fprintf(stderr, "Allocating %lld bytes to device(%d)...\n", N_bytes, loc);
		hipSetDevice(loc);
		*ptr = gpu_malloc(N_bytes);
	}
	cudaCheckErrors();
}

/// Free the CUDA device  memory pointed by 'gpuptr' (+errorcheck)
void gpu_free(void *gpuptr) {
	massert(hipFree(gpuptr) == hipSuccess,
          hipGetErrorString(hipGetLastError()));
}

/// Free the CUDA pinned memory pointed by 'gpuptr' (+errorcheck)
void pin_free(void *gpuptr) {
	massert(hipHostFree(gpuptr) == hipSuccess,
          hipGetErrorString(hipGetLastError()));
}

void vec_free(void ** ptr, int loc){
	int count = 666;
	hipGetDeviceCount(&count);

	if (-2 == loc) free(*ptr);
	else if (-1 == loc || -3 == loc) pin_free(*ptr);
	else if (loc >= count || loc < 0) error("vec_free: Invalid device");
	else {
		hipSetDevice(loc);
		gpu_free(*ptr);
	}
	cudaCheckErrors();
}

void Sinit_cuRAND(float * dev_ptr, long long length, int seed){
    	hiprandGenerator_t gen;
	/* Create pseudo-random number generator */
	massert(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) == hipSuccess,
          hipGetErrorString(hipGetLastError()));
    
	/* Set seed */
	massert(hiprandSetPseudoRandomGeneratorSeed(gen, seed) == hipSuccess,
          hipGetErrorString(hipGetLastError()));

	/* Generate length floats on device */
	massert(hiprandGenerateUniform(gen, dev_ptr, length) == hipSuccess,
          hipGetErrorString(hipGetLastError()));
}

void Dinit_cuRAND(double * dev_ptr, long long length, int seed){
    	hiprandGenerator_t gen;
	/* Create pseudo-random number generator */
	massert(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) == hipSuccess,
          hipGetErrorString(hipGetLastError()));
    
	/* Set seed */
	massert(hiprandSetPseudoRandomGeneratorSeed(gen, seed) == hipSuccess,
          hipGetErrorString(hipGetLastError()));

	/* Generate length doubles on device */
	massert(hiprandGenerateUniformDouble(gen, dev_ptr, length) == hipSuccess,
          hipGetErrorString(hipGetLastError()));
}
