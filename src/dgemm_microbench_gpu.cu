#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A hipblasDgemm micro-benchmark
///

#include <cassert>
#include "cpu_utils.hpp"
#include "gpu_utils.hpp"

char* check_benchmark(short dev_id, size_t minDim, size_t Mmax, size_t Nmax, size_t Kmax, size_t M_step, size_t N_step, size_t K_step, size_t samples_max){
	char *filename = (char *) malloc(256* sizeof(char));
	sprintf(filename, "%s/BenchOutputs/%s/hipblasDgemm/EMLR_dev-%d_minDim-%d_max-%d-%d-%d_step-%d-%d-%d_lim-%d.log", PROJECTDIR, MACHINE, dev_id, minDim, Mmax, Nmax, Kmax, M_step, N_step, K_step, samples_max);

	FILE* fp = fopen(filename,"r");
	if (!fp) { 
		fp = fopen(filename,"w+");
		if (!fp) error("report_results: LogFile failed to open");
		else warning("Generating Logfile...");
		fclose(fp);
	}
	else {
		fprintf(stderr,"GPU DGEMM benchmark@%s dev=%d found: minDim = %d, max dims(%d,%d,%d) with steps(%d,%d,%d) and max %d benchmarks\n", MACHINE, dev_id, minDim, Mmax, Nmax, Kmax, M_step, N_step, K_step, samples_max);
		fclose(fp);	
		exit(1); 
	}
	return filename;		  	
}

void report_run(char* filename, short dev_id, size_t M, size_t N, size_t K, double cublas_t_av, double cublas_t_min, double cublas_t_max){

	FILE* fp = fopen(filename,"a");
	if (!fp) error("report_run: LogFile failed to open");
   	fprintf(fp,"%d,%d,%d, %e,%e,%e\n", M, N, K, cublas_t_av, cublas_t_min, cublas_t_max);
        fclose(fp); 
}

int main(const int argc, const char *argv[]) {

  	double alpha, beta;
  	alpha = 1.1234, beta = 1.2345;

  	int ctr = 1, dev_id, samples_max, bench_num = 0;

	char machine[256];
	size_t minDim, Mmax, Nmax, Kmax;
	size_t M_step, N_step , K_step;

	switch (argc) {
	case (11):
		sprintf(machine , "%s", argv[ctr++]);
		dev_id = atoi(argv[ctr++]);
		minDim = atoi(argv[ctr++]);
		Mmax = atoi(argv[ctr++]);
		Nmax = atoi(argv[ctr++]);
		Kmax = atoi(argv[ctr++]);
		M_step = atoi(argv[ctr++]);
		N_step = atoi(argv[ctr++]);
		K_step = atoi(argv[ctr++]);
		samples_max = atoi(argv[ctr++]);
		break;
	default:
		error("Incorrect input arguments. Usage: ./correct_run machine dev_id minDim Mmax Nmax Kmax M_step N_step K_step max_benchmarks");
  	}

	if (strcmp(MACHINE, machine)) error("dgemm_microbench_gpu: Running on wrong machine");
	char *filename = check_benchmark(dev_id, minDim, Mmax, Nmax, Kmax, M_step, N_step, K_step, samples_max);

	/// Matrix Layouts for GPU GEMM
	hipblasOperation_t gpu_op_A = HIPBLAS_OP_N, gpu_op_B = HIPBLAS_OP_N;  // HIPBLAS_OP_N, HIPBLAS_OP_T

	size_t ldA = Mmax, ldB = Kmax, ldC = Mmax;

	/// Set device 
	hipSetDevice(dev_id);

	hipblasHandle_t handle0;
 	hipStream_t host_stream;

  	hipStreamCreate(&host_stream);
	assert(HIPBLAS_STATUS_SUCCESS == hipblasCreate(&handle0));
	assert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(handle0, host_stream));

	double cpu_timer = csecond();

	double *A_dev, *B_dev, *C_dev;
  	vec_alloc((void**)&A_dev, Mmax * Kmax * sizeof(double), dev_id);
  	vec_alloc((void**)&B_dev, Nmax * Kmax * sizeof(double), dev_id);
  	vec_alloc((void**)&C_dev, Mmax * Nmax * sizeof(double), dev_id);
	cudaCheckErrors();

	cpu_timer  = csecond() - cpu_timer ;
	fprintf(stderr, "Allocated Device memory t_aloc = %lf ms\n", cpu_timer  * 1000);

	cpu_timer = csecond();

	Dinit_cuRAND(A_dev, Mmax * Kmax, 42);
	Dinit_cuRAND(B_dev, Kmax * Nmax, 42*42);
	Dinit_cuRAND(C_dev, Mmax * Nmax, 42*42*42);

	cudaCheckErrors();
	cpu_timer  = csecond() - cpu_timer ;	
	fprintf(stderr, "Initialized Device Mem: t_init = %lf ms\n", cpu_timer  * 1000);

	fprintf(stderr, "\nMatrix details: A(%s) B(%s) C(%s) -> Mmax = %d, Nmax = %d, Kmax = %d\n",
            print_mem(COL_MAJOR), print_mem(COL_MAJOR), print_mem(COL_MAJOR), Mmax, Nmax, Kmax);
	fprintf(stderr, "Constants: alpha = %lf, beta = %lf\n", alpha, beta);

	// Warmup 
	for ( int itt = 0; itt <10; itt++){
		assert(HIPBLAS_STATUS_SUCCESS == hipblasDgemm(handle0, gpu_op_A, gpu_op_B, Mmax, Nmax, Kmax, &alpha, A_dev, ldA, B_dev, ldB, &beta, C_dev, ldC));
		hipStreamSynchronize(host_stream);
	}
	cudaCheckErrors();
	double cublas_t_av, cublas_t_min , cublas_t_max; 
	size_t bench_ctr = 0;
	for (size_t M = minDim; M < Mmax + 1; M+=M_step) 
		for (size_t N = minDim; N < Nmax + 1; N+=N_step) 
			for (size_t K = minDim; K < Kmax + 1; K+=K_step) {
					ldA = M, ldB = K, ldC = M;
					// TODO: Special case for square tilling : Non-equal N,M not of great interest
					if(M != N || bench_ctr > samples_max) break; 
					fprintf(stderr,"Running CUBLAS GPU-> M = %d, N = %d, K = %d\n", M, N, K);
					cublas_t_av = cublas_t_max = 0;
					cublas_t_min = 1e9;
					for (int itt = 0; itt < ITER; itt ++) {
						cpu_timer = csecond();
						assert(HIPBLAS_STATUS_SUCCESS == hipblasDgemm(handle0, gpu_op_A, gpu_op_B, M, N, K, &alpha, A_dev, ldA, B_dev, ldB, &beta, C_dev, ldC));
						hipStreamSynchronize(host_stream);
						cpu_timer  = csecond() - cpu_timer ;
						cublas_t_av += cpu_timer;
						if (cpu_timer > cublas_t_max) cublas_t_max = cpu_timer; 
						if (cpu_timer < cublas_t_min) cublas_t_min = cpu_timer; 
					}
					cublas_t_av /= ITER;
					fprintf(stderr, "CUBLAS GPU:\n GPU exec time:\t Average=%lf ms, Min = %lf ms, Max = %lf ms\n", cublas_t_av  * 1000, cublas_t_min  * 1000, cublas_t_max  * 1000);
					cudaCheckErrors();

					report_run(filename, dev_id, M, N, K, cublas_t_av, cublas_t_min, cublas_t_max); 
					bench_ctr++;
			}
	fprintf(stderr, "Ran %d Benchmarks.Finallizing...\n", bench_ctr);
	return 0;
}
